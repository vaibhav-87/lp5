#include "hip/hip_runtime.h"
#include <math.h>
#include <time.h>
#include <iostream>
#include "hip/hip_runtime.h"

void cpuSum(int *A, int *B, int *C, int N)
{
    for (int i = 0; i < N; ++i)
    {
        C[i] = A[i] + B[i];
    }
}

__global__ void kernel(int *A, int *B, int *C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

void gpuSum(int *A, int *B, int *C, int N)
{
    int threadsPerBlock = min(1024, N);
    int blocksPerGrid = ceil(double(N) / double(threadsPerBlock));
    kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
}

bool isVectorEqual(int *A, int *B, int N)
{
    for (int i = 0; i < N; ++i)
    {
        if (A[i] != B[i])
        {
            return false;
        }
    }
    return true;
}

int main()
{
    int N = 2e7;
    int *A, *B, *C, *D, *a, *b, *c;
    int size = N * sizeof(int);

    A = (int *)malloc(size);
    B = (int *)malloc(size);
    C = (int *)malloc(size);
    D = (int *)malloc(size);

    for (int i = 0; i < N; ++i)
    {
        A[i] = rand() % 1000;
        B[i] = rand() % 1000;
    }

    clock_t start, end;

    start = clock();
    cpuSum(A, B, C, N);
    end = clock();
    float timeTakenCPU = ((float)(end - start)) / CLOCKS_PER_SEC;

    hipMalloc(&a, size);
    hipMalloc(&b, size);
    hipMalloc(&c, size);

    hipMemcpy(a, A, size, hipMemcpyHostToDevice);
    hipMemcpy(b, B, size, hipMemcpyHostToDevice);

    start = clock();
    gpuSum(a, b, c, N);
    hipDeviceSynchronize();
    hipMemcpy(D, c, size, hipMemcpyDeviceToHost);
    end = clock();
    float timeTakenGPU = ((float)(end - start)) / CLOCKS_PER_SEC;

    hipFree(a);
    hipFree(b);
    hipFree(c);

    bool success = isVectorEqual(C, D, N);

    printf("Vector Addition\n");
    printf("--------------------\n");
    printf("CPU Time: %f \n", timeTakenCPU);
    printf("GPU Time: %f \n", timeTakenGPU);
    printf("Speed Up: %f \n", timeTakenCPU / timeTakenGPU);
    printf("Verification: %s \n", success ? "true" : "false");
}
